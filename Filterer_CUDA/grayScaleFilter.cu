#include "hip/hip_runtime.h"
#include <stdio.h>
#include "lodepng.h"

// kernel run on each thread
__global__ void grayScaleFilter(unsigned char * d_out, unsigned char * d_in){
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int rgbAverage = 0;
  int sum = 0;
  for (int i = 0; i < 3; i++) { 
    sum = sum + d_in[4*idx+i];
  }
  rgbAverage = sum/3;
  for (int i = 0; i < 3; i++) { // for RGB channels, apply grayScaleFilter
    d_out[4*idx+i] = rgbAverage;
  }
  d_out[4*idx + 3] = d_in[4*idx + 3]; // keep the same value for alpha channel
}

int main(int argc, char ** argv) {
  char * in_filename = argv[1];
  char * out_filename = argv[2];
  printf("Applying grayscale filter on %s\n", in_filename);

  // load input image 
  unsigned error;
  unsigned char *h_in_img, *h_out_img, *d_in_img, *d_out_img;
  unsigned width, height;

  // error handling
  error = lodepng_decode32_file(&h_in_img, &width, &height, in_filename);
  if (error) printf("error %u: %s\n", error, lodepng_error_text(error));

  const int IMAGE_PIXELS = width*height;
  const int IMAGE_BYTES = IMAGE_PIXELS * 4 * sizeof(unsigned char);
  const int THREADS_PER_BLOCK = 1024;
  const int BLOCK_COUNT = IMAGE_PIXELS/THREADS_PER_BLOCK;
  
  // allocate CPU memory
  h_out_img = (unsigned char *) malloc(IMAGE_BYTES);

  // allocate GPU memory
  hipMalloc(&d_in_img, IMAGE_BYTES);
  hipMalloc(&d_out_img, IMAGE_BYTES);

  // transfer image to GPU
  hipMemcpy(d_in_img, h_in_img, IMAGE_BYTES, hipMemcpyHostToDevice);

  // launch kernel
  grayScaleFilter<<<BLOCK_COUNT, THREADS_PER_BLOCK>>>(d_out_img, d_in_img);
  
  // copy back result array to CPU
  hipMemcpy(h_out_img, d_out_img, IMAGE_BYTES, hipMemcpyDeviceToHost);

  int rgbAverage = 0;
  int sum = 0;

  // process remainder on CPU since image size not always evenly divisible by block size
  int remainder = IMAGE_PIXELS % THREADS_PER_BLOCK;
  for (int idx = IMAGE_PIXELS - remainder; idx < IMAGE_PIXELS; idx++) {
    for (int i = 0; i < 3; i++) { // for RGB channels, apply grayScaleFilter
      sum = sum + h_in_img[4*idx+i];
    }
  	h_out_img[4*idx + 3] = h_in_img[4*idx + 3]; // keep the same value for alpha channel
  }

  rgbAverage = sum/3;
  for (int idx = IMAGE_PIXELS - remainder; idx < IMAGE_PIXELS; idx++) {
    for (int i = 0; i < 3; i++) { // for RGB channels, apply grayScaleFilter
       h_out_img[4*idx + i] = rgbAverage; // keep the same value for alpha channel
    }
  }

  // save output image 
  lodepng_encode32_file(out_filename, h_out_img, width, height);

  // cleanup
  hipFree(d_in_img);
  hipFree(d_out_img);
  free(h_in_img);
  free(h_out_img);

  return 0;
}